﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;  

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;        // 4096 volume[0-4095]
  fs->FCB_SIZE = FCB_SIZE;                      // 32, name-20, created-4, modified-4, start-2, size-2
  fs->FCB_ENTRIES = FCB_ENTRIES;                // 1024, total size 32768, volume[4096-36863]
  fs->STORAGE_SIZE = VOLUME_SIZE;               // 1085440
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;  // 32, total 2^15 blocks, volume[36864-1085440]
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;    // 20
  fs->MAX_FILE_NUM = MAX_FILE_NUM;              // 1024
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;            // 1048576
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;    // 36864

  // initialize the Super Block
  for (int i = 0; i < SUPERBLOCK_SIZE; ++ i){
    volume[i] = 255;  // 255 means 1111 1111, each uchar represents 8-blocks bit map 
  }
}



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{ 
	//printf("fs_open was called\n");
  gtime += 1; // increment the timeing



  /* Implement open operation here */
  int i, j;
  int index = -1;
  bool is_match = true;
  u32 base;

  // serch the file info in FCB
  for (i = fs->SUPERBLOCK_SIZE; i < fs->FILE_BASE_ADDRESS; i += fs->FCB_SIZE){
	  is_match = true;
	  for (j = 0; j < fs->MAX_FILENAME_SIZE; ++j) {
		  if (s[j] != fs->volume[i + j]) {
			  is_match = false;
			  break;
		  }
	  }
	if (is_match) {
      index = i;
      break;
    }
  }
  if (index != -1) {    // the file exists

    // update the modified time
    fs->volume[index + 31] = gtime % 256;
    fs->volume[index + 30] = (gtime >> 8) % 256;
    u32 address = fs->volume[index+23] + (fs->volume[index+22]<<8) + (fs->volume[index+21]<<16) + (fs->volume[index+20]<<24);
    return address;

  }else{                // the file does not exist

    // create a new zero byte file
    if (op == 1) {      
      // serch for free blocks in the Super Block
      for (i = 0; i < fs->SUPERBLOCK_SIZE; i += 4){
        if (fs->volume[i] == 255){  // every uchar represents 8 bits(block)
          fs->volume[i] = 127;
          index = i * 8;    // #start block of the file
          break;
        }
      } // end for

      // add the info of new file into FCB
      base;     // start physical address of the target FCB 
      u32 created, modified, start, size;   // the file infomation

      base = fs->SUPERBLOCK_SIZE + index;  // 4096 + (index / 32) * 32
      start = fs->FILE_BASE_ADDRESS + index * (fs->STORAGE_BLOCK_SIZE);  // the physical address of the file in byte
      size = 0;           // the size of the file in in byte
      created = gtime;    // get current time as created time
      modified = gtime;   // get current time as modified time
      

      // store the file name
	  for (j = 0; j < fs->MAX_FILENAME_SIZE; ++j) {
		  fs->volume[base + j] = s[j];
		  if (s[j] == '\0')
			  break;
	  }

      // store the physical start address of the file (4B)
      fs->volume[base + 23] = start % 256;
      fs->volume[base + 22] = (start >> 8) % 256;
      fs->volume[base + 21] = (start >> 16) % 256;
      fs->volume[base + 20] = (start >> 24) % 256;

      // store the size of the file (4B)
      fs->volume[base + 27] = size % 256;
      fs->volume[base + 26] = (size >> 8) % 256;
      fs->volume[base + 25] = (size >> 16) % 256;
      fs->volume[base + 24] = (size >> 24) % 256;

      // store the created time 
      fs->volume[base + 29] = created % 256;
      fs->volume[base + 28] = (created >> 8) % 256;

      // store the modified time
      fs->volume[base + 31] = modified % 256;
      fs->volume[base + 30] = (modified >> 8) % 256;

      return start;
    } // end if

  } // end else
  
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
  /* Implement read operation here */
  // gtime += 1;
	//printf("fs_read was called\n");
  for (int i = 0; i < size; ++ i) {
    output[i] = (char)fs->volume[fp++];
  }

}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{	
	gtime++;

  /* Implement write operation here */
  u32 index, base, file_size;
  index = (fp - fs->FILE_BASE_ADDRESS) >> 10;   // the index of the entry of FCB
  base = fs->SUPERBLOCK_SIZE + index * fs->FCB_SIZE;

  // update the modified time
  fs->volume[base+31] = gtime % 256;
  fs->volume[base+30] = (gtime >> 8) % 256;

  // get the size of older file
  file_size = fs->volume[base+27] + (fs->volume[base+26]<<8) + (fs->volume[base+25]<<16) + (fs->volume[base+24]<<24);

  // update the size of file
  fs->volume[base + 27] = size % 256;
  fs->volume[base + 26] = (size >> 8) % 256;

  // write the physical storage
  for (int i = 0; i < size; ++ i){

    // update the super block everytime writing a new block
    if (i % 32 == 0) {
      int block_order = i / 32;
      uchar target = fs->volume[index * 4 + block_order / 8];
	  if (target / (1 << (7 - (block_order % 8))) % 2 == 1)
		target -= (1 << (7 - (block_order % 8)));         // target block: 1 -> 0
      fs->volume[index * 4 + block_order / 8] = target;
    }
    // write
    fs->volume[fp++] = input[i]; 
  }

  // cleanup the older contents left
  if (file_size > size) {
    for (int i = fp; i < fp + file_size - size; ++i){
      fs->volume[i] = '\0';
    }
  }

  return fp;

}
__device__ void fs_gsys(FileSystem *fs, int op)
{
  /* Implement LS_D and LS_S operation here */

  // data structure to sort the modified time
  // the element of sort_arr is 4 bytes 
  // the first two bytes represent the FCB entry
  // the last two bytes represent the modified time
  u32 sort_arr[1024];
  int arr_size = 0;                                   // track the size of array

  u32 base, modified, size, entry;
  // LS_D: file name, order by modified time
  if (op == 0) {
    for (int i = 0; i < fs->FCB_ENTRIES; ++ i){
      base = fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE;
      // FCB entry is not empty
      if (fs->volume[base] != '\0'){
        entry = i;
        modified = fs->volume[base+31] + (fs->volume[base+30] << 8);
		    int ptr = arr_size - 1;
        sort_arr[arr_size++] = (entry << 16) + modified; // add to the sort array
        // insertion sort
        while (ptr >= 0 && modified < sort_arr[ptr] % (1 << 16)) {
          sort_arr[ptr + 1] = sort_arr[ptr];
          sort_arr[ptr] = (entry << 16) + modified;
          ptr--;
        }	// end while
      }	// end if
    } // end for

	// print information
	printf("===sort by modified time===\n");
	while (--arr_size >= 0) {
		entry = sort_arr[arr_size] >> 16; 
		printf("%s\n", &fs->volume[fs->SUPERBLOCK_SIZE + entry * fs->FCB_SIZE]);
	}

  }	// end if

  // LS_S: file name and size, order by size
  if (op == 1) {
	  for (int i = 0; i < fs->FCB_ENTRIES; ++i) {
		  base = fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE;
		  // FCB entry is not empty
		  if (fs->volume[base] != '\0') {
			  entry = i;
			  size = fs->volume[base + 27] + (fs->volume[base + 26] << 8);
			  int ptr = arr_size - 1;
			  sort_arr[arr_size++] = (entry << 16) + size; // add to the sort array
			  // insertion sort
			  while (ptr >= 0 && size <= sort_arr[ptr] % (1 << 16)) {
				  sort_arr[ptr + 1] = sort_arr[ptr];
				  sort_arr[ptr] = (entry << 16) + size;
				  ptr--;
			  }	// end while
		  }	// end if
	  } // end for

	  // print information
	  printf("===sort by file size===\n");
	  while (--arr_size >= 0) {
		  entry = sort_arr[arr_size] >> 16;
		  base = fs->SUPERBLOCK_SIZE + entry * fs->FCB_SIZE;
		  size = sort_arr[arr_size] % (1 << 16);
		  printf("%s %d\n", &fs->volume[base], size);
	  }

  }	// end if
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
	int i, j;
	int index = -1;
	bool is_match;
	if (op == 2) {
		for (i = fs->SUPERBLOCK_SIZE; i < fs->FILE_BASE_ADDRESS; i += fs->FCB_SIZE) {
			is_match = true;
			for (j = 0; j < fs->MAX_FILENAME_SIZE; ++j) {
				if (s[j] != fs->volume[i + j]) {
					is_match = false;
					break;
				}
			}
			if (is_match) {
				index = i;
				break;
			}
		} // end for
		if (index == -1) {
			printf("Cannot find file %s, please check!\n", s);
		}
		else {
			u32 address = fs->volume[index + 23] + (fs->volume[index + 22] << 8) + (fs->volume[index + 21] << 16) + (fs->volume[index + 20] << 24);
			u32 size = fs->volume[index + 27] + (fs->volume[index + 26] << 8);

			// release the file space
			for (i = 0; i < size; ++i) {
				fs->volume[address + i] = '\0';
			}
			// release the corresponding FCB
			for (i = 0; i < fs->FCB_SIZE; ++i) {
				fs->volume[index + i] = '\0';
			}
			// update the super block
			index = (index - fs->SUPERBLOCK_SIZE) / fs->FCB_SIZE;
			for (i = 0; i < 4; ++i) {
				fs->volume[index * 4 + i] = 255;
			}
		}
	}


}
