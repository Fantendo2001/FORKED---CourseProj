﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>




__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
	vm->invert_page_table[i + vm->PAGE_ENTRIES] = 0;//valid bit 0 means can be used
	vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = -2;	// -2 indicates that this line has not been used yet
	vm->invert_page_table[i + 3 * vm->PAGE_ENTRIES] = -2;	// -2 indicates that this line has not been used yet
  }
}


__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;
  vm->currenTime = 0;
  vm->storageCounter = 0;


  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  /* Complete vm_read function to read single element from data buffer */
	int offset = addr % 32;
	int base = addr >> 5;
	int target_index, head_index, tail_index;
	target_index = head_index = tail_index = -1;
	int left, right;
	for (int i = 0; i <vm->PAGE_ENTRIES; ++i){
		if (target_index < 0 && vm->invert_page_table[i] == base) target_index = i;	// find the corresponding index
		if (head_index < 0 && vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] == -1) head_index = i;	// find the most recent index
		if (tail_index < 0 && vm->invert_page_table[i + 3 * vm->PAGE_ENTRIES] == -1) tail_index = i;	// find the least recent index
		if (target_index >= 0 && head_index >= 0) break;	// break if the necessary indexes have been found
	}
	if (target_index >= 0) {
		// change the linked-list to refresh the priority
		left = vm->invert_page_table[target_index + 2 * vm->PAGE_ENTRIES];
		right = vm->invert_page_table[target_index + 3 * vm->PAGE_ENTRIES];
		if (left != -1) {		// target_index is not the head_index
			vm->invert_page_table[left + 3 * vm->PAGE_ENTRIES] = right;	// left.next = right
			vm->invert_page_table[target_index + 2 * vm->PAGE_ENTRIES] = -1;	// target.previous = -1
			if (right != -1) 		// target_index is not the last 
				vm->invert_page_table[right + 2 * vm->PAGE_ENTRIES] = left;	// right.previous = left
			vm->invert_page_table[head_index + 2 * vm->PAGE_ENTRIES] = target_index;	// head.previous = target
			vm->invert_page_table[target_index + 3 * vm->PAGE_ENTRIES] = head_index;	// target.next = head
		}
		printf("Read: %d; page faults: %d\n", (int)addr, vm->pagefault_num_ptr[0]);
		return vm->buffer[target_index * vm->PAGESIZE + offset];
	}
	else {	// page fault happens
		vm->pagefault_num_ptr[0] += 1;	// count the page fault
		// page swap
		int new_tail = vm->invert_page_table[tail_index + 2 * vm->PAGE_ENTRIES];
		// swap out
		for (int i = 0; i < vm->PAGESIZE; ++ i) {
			vm->storage[vm->invert_page_table[tail_index] * vm->PAGESIZE + i] = vm->buffer[tail_index * vm->PAGESIZE + i]; 
		}
		// swap in
		for (int i = 0; i < vm->PAGESIZE; ++ i) {
			vm->buffer[tail_index * vm->PAGESIZE + i] =  vm->storage[base * vm->PAGESIZE + i];
		}
		
		vm->invert_page_table[tail_index] = base;	// edit the new index of the page table
		vm->invert_page_table[new_tail + 3 * vm->PAGE_ENTRIES] = -1;	// new_tail.next = -1
		vm->invert_page_table[tail_index + 2 * vm->PAGE_ENTRIES] = -1;
		vm->invert_page_table[tail_index + 3 * vm->PAGE_ENTRIES] = head_index;
		vm->invert_page_table[head_index + 2 * vm->PAGE_ENTRIES] = tail_index;
		printf("Read: %d; page faults: %d\n", (int)addr, vm->pagefault_num_ptr[0]);
		return vm->buffer[tail_index * vm->PAGESIZE + offset];
	}

  return 123; //TODO
}


__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
	// vm: virtual memory
	// addr: the order of the input value, the last 5-bit is the offset and the remain 27-bit is the base
	int offset = addr % 32;
	int base = addr >> 5;
	int target_index, head_index, empty_index, tail_index;
	int left, right;
	target_index = head_index = empty_index = tail_index = -1;

	// get the index needed
	for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
		if (target_index < 0 && vm->invert_page_table[i] == base && vm->invert_page_table[i + vm->PAGE_ENTRIES] == 1)
			target_index = i;		// find the corresponding index
		if (head_index < 0 && vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] == -1) head_index = i;	// find the most recent index
		if (tail_index < 0 && vm->invert_page_table[i + 3 * vm->PAGE_ENTRIES] == -1) tail_index = i;	// find the least recent index
		if (empty_index < 0 && vm->invert_page_table[i] == 0x80000000) empty_index = i;	// find the empty index
		if (target_index >= 0 && head_index >= 0) break;	// break if the necessary indexes have been found
	}

	if (target_index >= 0) {
		vm->buffer[target_index * vm->PAGESIZE + offset] = value;
		// change the linked-list to refresh the priority
		left = vm->invert_page_table[target_index + 2 * vm->PAGE_ENTRIES];
		right = vm->invert_page_table[target_index + 3 * vm->PAGE_ENTRIES];
		if (left != -1) {		// target_index is not the head_index
			vm->invert_page_table[left + 3 * vm->PAGE_ENTRIES] = right;	// left.next = right
			vm->invert_page_table[target_index + 2 * vm->PAGE_ENTRIES] = -1;	// target.previous = -1
			if (right != -1) 		// target_index is not the last 
				vm->invert_page_table[right + 2 * vm->PAGE_ENTRIES] = left;	// right.previous = left
			vm->invert_page_table[head_index + 2 * vm->PAGE_ENTRIES] = target_index;	// head.previous = target
			vm->invert_page_table[target_index + 3 * vm->PAGE_ENTRIES] = head_index;	// target.next = head
		}
		printf("write: %d; page faults: %d\n", (int)addr, vm->pagefault_num_ptr[0]);
	}
	else {	// page fault happens
		vm->pagefault_num_ptr[0] += 1;	// count the page fault
		if (empty_index >= 0) {		// the page table is not empty
			vm->invert_page_table[empty_index] = base;	// edit the new index of the page table
			vm->invert_page_table[empty_index + vm->PAGE_ENTRIES] = 1;	// change the status
			vm->buffer[empty_index * vm->PAGESIZE + offset] = value;	// write into buffer
			printf("write: %d; page faults: %d\n", (int)addr, vm->pagefault_num_ptr[0]);
			if (head_index >= 0) {
				vm->invert_page_table[head_index + 2 * vm->PAGE_ENTRIES] = empty_index;
				vm->invert_page_table[empty_index + 2 * vm->PAGE_ENTRIES] = -1;
				vm->invert_page_table[empty_index + 3 * vm->PAGE_ENTRIES] = head_index;
			}
			else {
				vm->invert_page_table[empty_index + 2 * vm->PAGE_ENTRIES] = -1;
				vm->invert_page_table[empty_index + 3 * vm->PAGE_ENTRIES] = -1;
			}
		}
		else {						// the page table is full -- page swap
			int new_tail = vm->invert_page_table[tail_index + 2 * vm->PAGE_ENTRIES];
			// swap out
			for (int i = 0; i < vm->PAGESIZE; ++ i) {
				vm->storage[vm->invert_page_table[tail_index] * vm->PAGESIZE + i] = vm->buffer[tail_index * vm->PAGESIZE + i]; 
			}
			vm->buffer[tail_index * vm->PAGESIZE + offset] = value;
			printf("write: %d; page faults: %d\n", (int)addr, vm->pagefault_num_ptr[0]);
			vm->invert_page_table[tail_index] = base;	// edit the new index of the page table
			vm->invert_page_table[new_tail + 3 * vm->PAGE_ENTRIES] = -1;	// new_tail.next = -1
			vm->invert_page_table[tail_index + 2 * vm->PAGE_ENTRIES] = -1;
			vm->invert_page_table[tail_index + 3 * vm->PAGE_ENTRIES] = head_index;
			vm->invert_page_table[head_index + 2 * vm->PAGE_ENTRIES] = tail_index;
		}
	}
}


__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
	printf("Start of snapshot:\n");
   for (int i = 0; i < input_size; ++i){
	   results[i] = vm_read(vm, i);
	   printf("In snapshot - ");
   }
}

